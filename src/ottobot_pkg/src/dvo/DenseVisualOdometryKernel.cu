#include "hip/hip_runtime.h"
#include <DenseVisualOdometryKernel.cuh>

/*------------------------------------   KERNELS   -----------------------------------*/

__global__ void step_kernel(const unsigned char* gray,
                            const unsigned char* gray_prev,
                            const unsigned char* depth_prev,
                            float* residuals,
                            const float T[4][4],
                            const float cam_mat[3][3],
                            const float scale,
                            const int width,
                            const int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    
    if(tidx<width && tidy<height)
    {
        residuals[tidy*width + tidx] = compute_residual(gray,
                                                        tidx,
                                                        tidy,
                                                        gray_prev[tidy*width + tidx],
                                                        depth_prev[tidy*width + tidx],
                                                        T,
                                                        cam_mat,
                                                        scale,
                                                        width,
                                                        height);
    }
}

/*------------------------------------  FUNCTIONS  -----------------------------------*/

/*
 Bilinear interpolation
 Arguments:
 ---------
     x: array of x-coordinates to interpolate
     y: array of y-coordinates to interpolate
     dest: array of interpolation results (same dims as x and y)
     src_img: Original image
     width: src_img width
     height: src_img height
     N: Nb of points (dim of x, y, and dest)
*/
__device__ float bilinear_interpolation(   const float x, 
                                            const float y, 
                                            const unsigned char* src_img,
                                            const int width,
                                            const int height)
{
    int x0 = (int) floorf(x), y0 = (int) floorf(y);
    int x1 = x0 + 1, y1 = y0 + 1;   
    
    // Clip coordinates
    x0 = max(0, min(x0, width-1));
    y0 = max(0, min(y0, height-1));
    x1 = max(0, min(x1, width-1));
    y1 = max(0, min(y1, height-1));

    int Ia = (int) src_img[y0*width + x0];
    int Ib = (int) src_img[y1*width + x0];
    int Ic = (int) src_img[y0*width + x1];
    int Id = (int) src_img[y1*width + x1];
    
    // Calculate weights
    float wa = (x1 - x)*(y1 - y);
    float wb = (x1 - x)*(y - y0);
    float wc = (x - x0)*(y1 - y);
    float wd = (x - x0)*(y - y0);

    float result = (float) (wa*Ia + wb*Ib + wc*Ic + wd*Id);

    return result;
}                                            

__device__ float compute_residual(const unsigned char* gray,
                                  const int tidx,
                                  const int tidy,
                                  const unsigned char gray_prev,
                                  const unsigned char depth_prev,
                                  const float T[4][4],
                                  const float cam_mat[3][3],
                                  const float scale,
                                  const int width,
                                  const int height)
{   
    // Camera matrix
    // [fx  0 cx]
    // [ 0 fy cy]
    // [ 0  0  1]
    float fx = cam_mat[0][0], fy = cam_mat[1][1], cx = cam_mat[0][2], cy = cam_mat[1][2];

    float z = (float) depth_prev; 
    if(z >= 0.0)
    {   
        // Deproject prev image into 3d space
        float x = z*(tidx - cx)/fx, y = z*(tidy - cy)/fy;

        // Transform point
        float x_new, y_new, z_new;
        x_new = T[0][0]*x + T[0][1]*y + T[0][2]*z + T[0][3];
        y_new = T[1][0]*x + T[1][1]*y + T[1][2]*z + T[1][3];
        z_new = T[2][0]*x + T[2][1]*y + T[2][2]*z + T[2][3];

        // Project transformed point into new image
        float x_warped = (fx*x_new/z_new) + cx, y_warped = (fy*y_new/z_new);

        if(!( isnan(x_warped) || isnan(y_warped) ))
        {
            // Compute residuals
            float gray_projected = bilinear_interpolation(x_warped, y_warped, gray, width, height);
            return gray_projected - (float) gray_prev;
        }
    }

    return 0.0;
}                                  

